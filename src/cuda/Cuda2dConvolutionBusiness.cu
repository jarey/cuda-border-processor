#include "hip/hip_runtime.h"
/*
 * Cuda2dConvolutionBusiness.cpp
 *
 *  Created on: 05/04/2012
 *      Author: jose
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cutil.h>
#include "Cuda2dConvolutionBusiness.cuh"

Cuda2dConvolutionBusiness::Cuda2dConvolutionBusiness() {
	// TODO Auto-generated constructor stub
}

Cuda2dConvolutionBusiness::~Cuda2dConvolutionBusiness() {
	// TODO Auto-generated destructor stub
}
int Cuda2dConvolutionBusiness::iDivUp(int a, int b) {
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Align a to nearest higher multiple of b
int Cuda2dConvolutionBusiness::iAlignUp(int a, int b) {
	return (a % b != 0) ? (a - a % b + b) : a;
}
//Fast integer multiplication macro
#define IMUL(a, b) __mul24(a, b)

//Input data texture reference
texture<float, 2, hipReadModeElementType> textureData;

////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolution2D(float *d_Result, float *d_Kernel11,
		float* d_Kernel21, int dataW, int dataH, int sqrtFilterTam) {
	const int ix = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int iy = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	const float x = (float) ix + 0.5f;
	const float y = (float) iy + 0.5f;

	if (ix < dataW && iy < dataH) {
		float sum1 = 0;
		float sum2 = 0;
		for (int k = 0; k < sqrtFilterTam; k++) {
			for (int l = 0; l < sqrtFilterTam; l++) {
				sum1 += tex2D(textureData, (x) + (-k), (y) + (-l))
						* d_Kernel11[l * sqrtFilterTam + k];
				sum1 += tex2D(textureData, (x) + (-k), (y) + (-l))
						* d_Kernel21[l * sqrtFilterTam + k];
			}
		}

		d_Result[IMUL(iy, dataW) + ix] = sqrt(sum1 * sum1 + sum2 * sum2);
	}
}

__global__ void convolution2DLaplace(float *d_Result, float* d_Kernel11,
		int dataW, int dataH, int sqrtFilterTam) {
	const int ix = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int iy = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	const float x = (float) ix + 0.5f;
	const float y = (float) iy + 0.5f;

	if (ix < dataW && iy < dataH) {
		float sum1 = 0;
		float sum2 = 0;
		for (int k = 0; k < sqrtFilterTam; k++) {
			for (int l = 0; l < sqrtFilterTam; l++) {
				sum1 += tex2D(textureData, (x) + (-k), (y) + (-l))
						* d_Kernel11[l * sqrtFilterTam + k];
			}
		}

		if (sum1 > 255) {
			sum1 = 255.0;
		} else if (sum1 < 0) {
			sum1 = 0;
		}

		d_Result[IMUL(iy, dataW) + ix] = sum1;
	}
}

__global__ void convolutionRowGaussGPU(float *d_Result, float* d_Kernel1,
		int dataW, int dataH, int KERNEL_RADIUS) {
	const int ix = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int iy = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	const float x = (float) ix + 0.5f;
	const float y = (float) iy + 0.5f;

	if (ix < dataW && iy < dataH) {
		float sum = 0;

		for (int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
			sum += tex2D(textureData, x + k, y) * d_Kernel1[KERNEL_RADIUS - k];

		d_Result[IMUL(iy, dataW) + ix] = sum;
	}
}

////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnGaussGPU(float *d_Result, float* d_Kernel1,
		int dataW, int dataH, int KERNEL_RADIUS) {
	const int ix = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int iy = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	const float x = (float) ix + 0.5f;
	const float y = (float) iy + 0.5f;

	if (ix < dataW && iy < dataH) {
		float sum = 0;

		for (int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
			sum += tex2D(textureData, x, y + k) * d_Kernel1[KERNEL_RADIUS - k];

		d_Result[IMUL(iy, dataW) + ix] = sum;
	}
}

////////////////////////////////////////////////////////////////////////////////
// GPU convolution
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////

//Reduce problem size to have reasonable emulation time


float* Cuda2dConvolutionBusiness::Convolve(float *imagen, int ancho, int alto,
		float *h_kernel1, float *h_kernel2, int tamFilter) {

	float *h_DataA, *h_ResultGPU, *h_Kernelx1, *h_Kernelx2;
	int DATA_SIZE1 = 0;
	int DATA_W1 = 0;
	int DATA_H1 = 0;
	unsigned int width1 = 0;
	unsigned int height1 = 0;
	////////////////////////////////////////////////////////////////////////////////
	// Kernel configuration
	////////////////////////////////////////////////////////////////////////////////
	printf("");
	int KERNEL_TAM1 = tamFilter;
	int KERNEL_SIZE1 = KERNEL_TAM1 * sizeof(float);
	float *d_Kernel11;
	float *d_Kernel21;

	hipArray *a_Data;

	hipChannelFormatDesc floatTex = hipCreateChannelDesc<float> ();

	float *d_Result;

	int i, x, y;

	int sqrtFilterTam = sqrt(tamFilter);

	DATA_W1 = ancho;
	DATA_H1 = alto;
	DATA_SIZE1 = DATA_W1 * DATA_H1 * sizeof(float);

	h_DataA = (float *) malloc(DATA_SIZE1);
	h_ResultGPU = (float *) malloc(DATA_SIZE1);
	h_Kernelx1 = (float *) malloc(KERNEL_SIZE1);
	h_Kernelx2 = (float *) malloc(KERNEL_SIZE1);
	CUDA_SAFE_CALL(hipMallocArray(&a_Data, &floatTex, DATA_W1, DATA_H1));
	CUDA_SAFE_CALL(hipMalloc((void **) &d_Result, DATA_SIZE1));
	CUDA_SAFE_CALL(hipMalloc((void **) &d_Kernel11, KERNEL_SIZE1));
	CUDA_SAFE_CALL(hipMalloc((void **) &d_Kernel21, KERNEL_SIZE1));

	memcpy(h_Kernelx1, h_kernel1, tamFilter * sizeof(float));
	memcpy(h_Kernelx2, h_kernel2, tamFilter * sizeof(float));

	h_DataA = imagen;

	CUDA_SAFE_CALL(
			hipMemcpy(d_Kernel11, h_Kernelx1, KERNEL_SIZE1,
					hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(
			hipMemcpy(d_Kernel21, h_Kernelx1, KERNEL_SIZE1,
					hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(
			hipMemcpyToArray(a_Data, 0, 0, h_DataA, DATA_SIZE1,
					hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipBindTextureToArray(textureData, a_Data));
	//el ancho del bloque debe ser multiplo del máximo de memorria coalescente en escritura
	//para escrituras coalescentes en convolutionRowGPU y convolutionColumnGPU
	dim3 threadBlock(16, 12);
	dim3 blockGrid(this->iDivUp(DATA_W1, threadBlock.x),
			this->iDivUp(DATA_H1, threadBlock.y));

	CUDA_SAFE_CALL( hipDeviceSynchronize());
	convolution2D<<<blockGrid, threadBlock>>>(
			d_Result,
			d_Kernel11,
			d_Kernel21,
			DATA_W1,
			DATA_H1,
			sqrtFilterTam
	);

	CUDA_SAFE_CALL(hipDeviceSynchronize());

	CUDA_SAFE_CALL(
			hipMemcpy(h_ResultGPU, d_Result, DATA_SIZE1,
					hipMemcpyDeviceToHost));

	CUDA_SAFE_CALL(hipUnbindTexture(textureData));
	CUDA_SAFE_CALL(hipFree(d_Result));
	CUDA_SAFE_CALL(hipFree(d_Kernel11));
	CUDA_SAFE_CALL(hipFree(d_Kernel21));
	CUDA_SAFE_CALL(hipFreeArray(a_Data));
	free(h_Kernelx1);
	free(h_Kernelx2);

	return h_ResultGPU;

}

float* Cuda2dConvolutionBusiness::ConvolveLaplace(float *imagen, int ancho,
		int alto, float *h_kernel1, int tamFilter) {

	float *h_DataA, *h_ResultGPU, *h_Kernelx1;
	int DATA_SIZE1 = 0;
	int DATA_W1 = 0;
	int DATA_H1 = 0;
	unsigned int width1 = 0;
	unsigned int height1 = 0;
	////////////////////////////////////////////////////////////////////////////////
	// Configuración
	////////////////////////////////////////////////////////////////////////////////
	printf("");
	int KERNEL_TAM1 = tamFilter;
	int KERNEL_SIZE1 = KERNEL_TAM1 * sizeof(float);
	float *d_Kernel11;

	hipArray *a_Data;

	hipChannelFormatDesc floatTex = hipCreateChannelDesc<float> ();

	float *d_Result;

	int i, x, y;

	int sqrtFilterTam = sqrt(tamFilter);

	DATA_W1 = ancho;
	DATA_H1 = alto;
	DATA_SIZE1 = DATA_W1 * DATA_H1 * sizeof(float);

	h_DataA = (float *) malloc(DATA_SIZE1);
	h_ResultGPU = (float *) malloc(DATA_SIZE1);
	h_Kernelx1 = (float *) malloc(KERNEL_SIZE1);
	CUDA_SAFE_CALL(hipMallocArray(&a_Data, &floatTex, DATA_W1, DATA_H1));
	CUDA_SAFE_CALL(hipMalloc((void **) &d_Result, DATA_SIZE1));
	CUDA_SAFE_CALL(hipMalloc((void **) &d_Kernel11, KERNEL_SIZE1));

	memcpy(h_Kernelx1, h_kernel1, tamFilter * sizeof(float));

	h_DataA = imagen;

	CUDA_SAFE_CALL(
			hipMemcpy(d_Kernel11, h_Kernelx1, KERNEL_SIZE1,
					hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(
			hipMemcpyToArray(a_Data, 0, 0, h_DataA, DATA_SIZE1,
					hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipBindTextureToArray(textureData, a_Data));

	//el ancho del bloque debe ser multiplo del máximo de memorria coalescente en escritura
	//para escrituras coalescentes en convolutionRowGPU y convolutionColumnGP
	dim3 threadBlock(16, 12);
	dim3 blockGrid(this->iDivUp(DATA_W1, threadBlock.x),
			this->iDivUp(DATA_H1, threadBlock.y));

	CUDA_SAFE_CALL( hipDeviceSynchronize());
	convolution2DLaplace<<<blockGrid, threadBlock>>>(
			d_Result,
			d_Kernel11,
			DATA_W1,
			DATA_H1,
			sqrtFilterTam
	);
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	CUDA_SAFE_CALL(
			hipMemcpy(h_ResultGPU, d_Result, DATA_SIZE1,
					hipMemcpyDeviceToHost));

	CUDA_SAFE_CALL(hipUnbindTexture(textureData));
	CUDA_SAFE_CALL(hipFree(d_Result));
	CUDA_SAFE_CALL(hipFree(d_Kernel11));
	CUDA_SAFE_CALL(hipFreeArray(a_Data));

	free(h_Kernelx1);

	return h_ResultGPU;
}

float* Cuda2dConvolutionBusiness::ConvolveGauss(float *imagen, int ancho,
		int alto, float *h_kernel1, int radiusFilter) {

	float *h_DataA, *h_ResultGPU, *h_Kernelx1;
	int DATA_SIZE1 = 0;
	int DATA_W1 = 0;
	int DATA_H1 = 0;
	unsigned int width1 = 0;
	unsigned int height1 = 0;
	////////////////////////////////////////////////////////////////////////////////
	// Configuración del kernel
	////////////////////////////////////////////////////////////////////////////////
	printf("");
	int KERNEL_TAM1 = radiusFilter;
	int KERNEL_SIZE1 = (2 * KERNEL_TAM1 + 1) * sizeof(float);
	float *d_Kernel11;

	hipArray *a_Data;
	hipChannelFormatDesc floatTex = hipCreateChannelDesc<float> ();
	float *d_Result;
	int i, x, y;
	DATA_W1 = ancho;
	DATA_H1 = alto;
	DATA_SIZE1 = DATA_W1 * DATA_H1 * sizeof(float);

	h_DataA = (float *) malloc(DATA_SIZE1);
	h_ResultGPU = (float *) malloc(DATA_SIZE1);
	h_Kernelx1 = (float *) malloc(KERNEL_SIZE1);
	CUDA_SAFE_CALL(hipMallocArray(&a_Data, &floatTex, DATA_W1, DATA_H1));
	CUDA_SAFE_CALL(hipMalloc((void **) &d_Result, DATA_SIZE1));
	CUDA_SAFE_CALL(hipMalloc((void **) &d_Kernel11, KERNEL_SIZE1));

	memcpy(h_Kernelx1, h_kernel1, KERNEL_SIZE1 * sizeof(float));

	h_DataA = imagen;

	CUDA_SAFE_CALL(
			hipMemcpy(d_Kernel11, h_Kernelx1, KERNEL_SIZE1,
					hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(
			hipMemcpyToArray(a_Data, 0, 0, h_DataA, DATA_SIZE1,
					hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipBindTextureToArray(textureData, a_Data));
	//el ancho del bloque debe ser multiplo del máximo de memorria coalescente en escritura
	//para escrituras coalescentes en convolutionRowGPU y convolutionColumnGP
	dim3 threadBlock(16, 12);
	dim3 blockGrid(this->iDivUp(DATA_W1, threadBlock.x),
			this->iDivUp(DATA_H1, threadBlock.y));

	CUDA_SAFE_CALL( hipDeviceSynchronize());
	convolutionRowGaussGPU<<<blockGrid, threadBlock>>>(
			d_Result,
			d_Kernel11,
			DATA_W1,
			DATA_H1,
			KERNEL_TAM1
	);
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	CUDA_SAFE_CALL(
			hipMemcpyToArray(a_Data, 0, 0, d_Result, DATA_SIZE1,
					hipMemcpyDeviceToDevice));
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	convolutionColumnGaussGPU<<<blockGrid, threadBlock>>>(
			d_Result,
			d_Kernel11,
			DATA_W1,
			DATA_H1,
			KERNEL_TAM1
	);

	CUDA_SAFE_CALL(hipDeviceSynchronize());

	CUDA_SAFE_CALL(
			hipMemcpy(h_ResultGPU, d_Result, DATA_SIZE1,
					hipMemcpyDeviceToHost));
;
	CUDA_SAFE_CALL(hipUnbindTexture(textureData));
	CUDA_SAFE_CALL(hipFree(d_Result));
	CUDA_SAFE_CALL(hipFree(d_Kernel11));
	CUDA_SAFE_CALL(hipFreeArray(a_Data));

	return h_ResultGPU;
}


////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRow(float *d_Result, float *d_kernel,
		int kernelRadius, int dataW, int dataH) {
	const int ix = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int iy = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	const float x = (float) ix + 0.5f;
	const float y = (float) iy + 0.5f;

	if (ix < dataW && iy < dataH) {
		float sum = 0;

		for (int k = -kernelRadius; k <= kernelRadius; k++) {
			sum += tex2D(textureData, x + k, y) * d_kernel[kernelRadius - k];
		}

		d_Result[IMUL(iy, dataW) + ix] = sum;
	}
}

////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumn(float *d_Result, float *d_kernel,
		int kernelRadius, int dataW, int dataH) {
	const int ix = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int iy = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	const float x = (float) ix + 0.5f;
	const float y = (float) iy + 0.5f;

	if (ix < dataW && iy < dataH) {
		float sum = 0;

		for (int k = -kernelRadius; k <= kernelRadius; k++) {
			sum += tex2D(textureData, x, y + k) * d_kernel[kernelRadius - k];
		}

		d_Result[IMUL(iy, dataW) + ix] = sum;
	}
}

////////////////////////////////////////////////////////////////////////////////
// GPU convolution
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////

//Reduce problem size to have reasonable emulation time


float* Cuda2dConvolutionBusiness::Convolution(float *imagen, int ancho,
		int alto, float *h_kernel1, float *h_kernel2, int kernelTam) {

	float *h_DataA, *h_ResultGPU, *h_Kernelx1, *h_Kernelx2;
	int DATA_SIZE1;
	int DATA_W1;
	int DATA_H1;
	////////////////////////////////////////////////////////////////////////////////
	// Kernel configuration
	////////////////////////////////////////////////////////////////////////////////
	printf("");
	int KERNEL_TAM1 = kernelTam;
	int KERNEL_SIZE1 = KERNEL_TAM1 * sizeof(float);
	float *d_Kernel11;
	float *d_Kernel21;

	hipArray *a_Data;

	hipChannelFormatDesc floatTex = hipCreateChannelDesc<float> ();

	float *d_Result;

	int kernelRadius = (kernelTam - 1) / 2;

	DATA_W1 = ancho;
	DATA_H1 = alto;
	DATA_SIZE1 = DATA_W1 * DATA_H1 * sizeof(float);

	h_DataA = (float *) malloc(DATA_SIZE1);
	h_ResultGPU = (float *) malloc(DATA_SIZE1);
	h_Kernelx1 = (float *) malloc(KERNEL_SIZE1);
	h_Kernelx2 = (float *) malloc(KERNEL_SIZE1);
	CUDA_SAFE_CALL(hipMallocArray(&a_Data, &floatTex, DATA_W1, DATA_H1));
	CUDA_SAFE_CALL(hipMalloc((void **) &d_Result, DATA_SIZE1));
	CUDA_SAFE_CALL(hipMalloc((void **) &d_Kernel11, KERNEL_SIZE1));
	CUDA_SAFE_CALL(hipMalloc((void **) &d_Kernel21, KERNEL_SIZE1));

	memcpy(h_Kernelx1, h_kernel1, kernelTam * sizeof(float));
	memcpy(h_Kernelx2, h_kernel2, kernelTam * sizeof(float));

	h_DataA = imagen;

	CUDA_SAFE_CALL(
			hipMemcpy(d_Kernel11, h_Kernelx1, KERNEL_SIZE1,
					hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(
			hipMemcpy(d_Kernel21, h_Kernelx1, KERNEL_SIZE1,
					hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(
			hipMemcpyToArray(a_Data, 0, 0, h_DataA, DATA_SIZE1,
					hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipBindTextureToArray(textureData, a_Data));
	//el ancho del bloque debe ser multiplo del máximo de memorria coalescente en escritura
	//para escrituras coalescentes en convolutionRowGPU y convolutionColumnGP
	dim3 threadBlock(16, 12);
	dim3 blockGrid(this->iDivUp(DATA_W1, threadBlock.x),
			this->iDivUp(DATA_H1, threadBlock.y));

	CUDA_SAFE_CALL( hipDeviceSynchronize());

	convolutionRow<<<blockGrid, threadBlock>>>(d_Result,d_Kernel11,kernelRadius, DATA_W1, DATA_H1);
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	CUDA_SAFE_CALL(
			hipMemcpyToArray(a_Data, 0, 0, d_Result, DATA_SIZE1,
					hipMemcpyDeviceToDevice));
	convolutionColumn<<<blockGrid, threadBlock>>>(d_Result,d_Kernel21,kernelRadius, DATA_W1, DATA_H1);
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	CUDA_SAFE_CALL(
			hipMemcpy(h_ResultGPU, d_Result, DATA_SIZE1,
					hipMemcpyDeviceToHost));

	CUDA_SAFE_CALL(hipUnbindTexture(textureData));
	CUDA_SAFE_CALL(hipFree(d_Result));
	CUDA_SAFE_CALL(hipFree(d_Kernel11));
	CUDA_SAFE_CALL(hipFree(d_Kernel21));
	CUDA_SAFE_CALL(hipFreeArray(a_Data));
	free(h_Kernelx1);
	free(h_Kernelx2);

	return h_ResultGPU;
}

